#include "hip/hip_runtime.h"
%%cu
//add two 2D matrix 
// here we launch one single thread in each block
// here blocks are arrange in 2D form


#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectorAdd(float *a,float *b, float *c){
    int y= blockIdx.y;
    int x= blockIdx.x;
    int id= gridDim.x*y+x;
    c[id]=a[id]+b[id];
    
}

int main(int *argc,char **argv[]){
    int columns=3;
    int rows=4;  

    float x[rows][columns]={{1,2,9},{3,4,9},{1,2,9},{3,4,9}};
    float y[rows][columns]={{1,2,9},{3,4,9},{1,2,9},{3,4,9}};
    float z[rows][columns];
    
    //variables for gpu
    float *d_x;
    float *d_y;
    float *d_z;

    //now we allocate space in gpu memory
    int array_size=rows*columns*sizeof(float);

    hipMalloc( (void**)&d_x,array_size);
    hipMalloc( (void**)&d_y,array_size);
    hipMalloc( (void**)&d_z,array_size);

    //now we copy these vector(array) in gpu memory
    hipMemcpy( d_x, x , array_size , hipMemcpyHostToDevice);
    hipMemcpy( d_y, y , array_size , hipMemcpyHostToDevice);

    // now we launch kernel
    dim3 blocks(rows,columns);
    vectorAdd<<<blocks,1>>>(d_x,d_y,d_z);

    // copy array from gpu to cpu memory
    hipMemcpy( z , d_z , array_size , hipMemcpyDeviceToHost);

    hipFree( d_x);
    hipFree( d_y);
    hipFree( d_z);


    // print the values of output array
    printf("values of array z:\n");
    for(int i=0;i<rows;i++){
        for(int j=0;j<columns;j++){
        printf(" %f",z[i][j]);
        }
        printf("\n");
    }
return 0;
}