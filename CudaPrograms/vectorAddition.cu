#include "hip/hip_runtime.h"
%%cu
#include<iostream>
#include<hip/hip_runtime.h>
using namespace std;

//function declaration
__global__ void vectorAdd(float*,float*,float*,int);


//function(kernel) Defination
__global__ void vectorAdd(float *A,float *B,float *C,int n){

    int id= threadIdx.x +blockDim.x*blockIdx.x;    // Id of particular thread:

    if(id<n)
        C[id]=A[id]+B[id];   
}

 //host program
 int main(){
   
    float h_A[]={1.1,2.2,3.3,4.4,5.5,6.6,7.7,8.8};
    float h_B[]={1.1,2.2,3.3,4.4,5.5,6.6,7.7,8.8};

    int n=sizeof(h_A)/sizeof(float);        // number of element in array
    float h_C[n];
    int size=sizeof(h_A);                   //size of array

    //gpu variables
    float *d_A,*d_B,*d_C;

    // memory allocation in gpu device
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);

    
    //host to device data memory transfer
    hipMemcpy(d_A ,h_A ,size , hipMemcpyHostToDevice);
    hipMemcpy(d_B ,h_B ,size , hipMemcpyHostToDevice);


    //number of threads and block launch:
    int threadPerBlock = 256;
    int blockPerGrid = (n+threadPerBlock-1)/threadPerBlock;

    //kernel launch:
    vectorAdd<<<blockPerGrid,threadPerBlock>>>(d_A,d_B,d_C,n); 

    
    //device to host memory transfer:
    hipMemcpy(h_C ,d_C ,size , hipMemcpyDeviceToHost);


    //deallocate the memory space in gpu device:
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    //traversal of output array:
    cout<<"output: ";
    for(int i=0;i<n;i++){
    cout<<h_C[i]<<" ";
    }
    return 0;
}