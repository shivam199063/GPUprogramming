#include "hip/hip_runtime.h"
%%cu
//Mul two 2D matrix 
// here we launch one single thread in each block


#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2
#define row2 4
#define col1 4
#define col2 2

__global__ void vectorMul(float *a,float *b, float *c){
    __shared__ float p[col1];
    int row = blockIdx.y;
    int col = blockIdx.x;
    int k = threadIdx.x;
    c[row*col2+col]=0;
    p[k]= a[row*col1+k]*b[k*col2+col];
    
    __syncthreads();
    for(int i=0;i<col1;i++){

        c[row*col2+col]+= p[i];
    }


}

int main(int *argc,char **argv[]){
    
    float x[row1][col1]={{1,2,3,4},{5,6,7,8}};
    float y[row2][col2]={{1,2},{3,4},{5,6},{7,8}};
    float z[row1][col2];
    
    //variables for gpu
    float *d_x;
    float *d_y;
    float *d_z;

    //now we allocate space in gpu memory
    int array_size=row1*col1*sizeof(float);
    int array_size_z=row1*col2*sizeof(float);

    hipMalloc( (void**)&d_x,array_size);
    hipMalloc( (void**)&d_y,array_size);
    hipMalloc( (void**)&d_z,array_size_z);

    //now we copy these vector(array) in gpu memory
    hipMemcpy( d_x, x , array_size , hipMemcpyHostToDevice);
    hipMemcpy( d_y, y , array_size , hipMemcpyHostToDevice);
    
    dim3 blocks(row1,col2,1);
    // now we launch kernel
    vectorMul<<<blocks,col1>>>(d_x,d_y,d_z);

    // copy array from gpu to cpu memory
    hipMemcpy( z , d_z , array_size_z , hipMemcpyDeviceToHost);

    hipFree( d_x);
    hipFree( d_y);
    hipFree( d_z);


    // print the values of output array
      printf("values of array z:\n");
    for(int i=0;i<row1;i++){
        for(int j=0;j<col2;j++){
        printf(" %f",z[i][j]);
        }
        printf("\n");
    }
return 0;
}