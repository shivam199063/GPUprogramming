#include "hip/hip_runtime.h"
%%cu
//Mul two 2D matrix 
// here we launch one single thread in each block


#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2
#define row2 4
#define col1 4
#define col2 2

__global__ void vectorMul(float *a,float *b, float *c){
    int row= blockIdx.y*blockDim.y+threadIdx.y;
    int col= blockIdx.x*blockDim.x+threadIdx.x;
    int width_a= col1;
    int width_b= col2;
    int width_c= col2;      
    float sum=0;
    for(int i=0;i<col1;i++){
    sum = sum + a[row*width_a+i]*b[i*width_b+col];
    }
    c[row*width_c+col] = sum;
}

int main(int *argc,char **argv[]){
    
    float x[row1][col1]={{1,2,3,4},{5,6,7,8}};
    float y[row2][col2]={{1,2},{3,4},{5,6},{7,8}};
    float z[row1][col2];
    
    //variables for gpu
    float *d_x;
    float *d_y;
    float *d_z;

    //now we allocate space in gpu memory
    int array_size=row1*col1*sizeof(float);
    int array_size_z=row1*col2*sizeof(float);

    hipMalloc( (void**)&d_x,array_size);
    hipMalloc( (void**)&d_y,array_size);
    hipMalloc( (void**)&d_z,array_size_z);

    //now we copy these vector(array) in gpu memory
    hipMemcpy( d_x, x , array_size , hipMemcpyHostToDevice);
    hipMemcpy( d_y, y , array_size , hipMemcpyHostToDevice);
    
    dim3 blocks(row1,col2,1);
    // now we launch kernel
    vectorMul<<<blocks,1>>>(d_x,d_y,d_z);

    // copy array from gpu to cpu memory
    hipMemcpy( z , d_z , array_size_z , hipMemcpyDeviceToHost);

    hipFree( d_x);
    hipFree( d_y);
    hipFree( d_z);


    // print the values of output array
      printf("values of array z:\n");
    for(int i=0;i<row1;i++){
        for(int j=0;j<col2;j++){
        printf(" %f",z[i][j]);
        }
        printf("\n");
    }
return 0;
}