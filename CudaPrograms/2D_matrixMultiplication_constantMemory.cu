#include "hip/hip_runtime.h"
%%cu
//Mul two 2D matrix 
// here we launch one single thread in each block


#include<stdio.h>
#include<hip/hip_runtime.h>
#define row1 2
#define row2 4
#define col1 4
#define col2 2

__constant__ float x[row1][col1];

__global__ void vectorMul(float *y, float *z){
    __shared__ int p[col1];
    int row = blockIdx.y;
    int col = blockIdx.x;
    int k = threadIdx.x;
    z[row*col2+col]=0;
    p[k]= x[row][k]*y[k*col2+col];
    
    __syncthreads();
    for(int i=0;i<col1;i++){

        z[row*col2+col]=z[row*col2+col]+p[i];
    }


}

int main(int *argc,char **argv[]){
    
    float c[row1][col1]={{1,2,3,4},{5,6,7,8}};
    float y[row2][col2]={{1,2},{3,4},{5,6},{7,8}};
    float z[row1][col2];
    
    //variables for gpu
    
    float *d_y;
    float *d_z;

    //now we allocate space in gpu memory
    int array_size=row1*col1*sizeof(float);
    int array_size_z=row1*col2*sizeof(float);

    
    hipMalloc( (void**)&d_y,array_size);
    hipMalloc( (void**)&d_z,array_size_z);

    //now we copy these vector(array) in gpu memory
    hipMemcpyToSymbol(HIP_SYMBOL( x), c, array_size);
    hipMemcpy( d_y, y , array_size , hipMemcpyHostToDevice);
    
    dim3 blocks(row1,col2,1);
    // now we launch kernel
    vectorMul<<<blocks,col1>>>(d_y,d_z);

    // copy array from gpu to cpu memory
    hipMemcpy( z , d_z , array_size_z , hipMemcpyDeviceToHost);

  
    hipFree( d_y);
    hipFree( d_z);


    // print the values of output array
      printf("values of array z:\n");
    for(int i=0;i<row1;i++){
        for(int j=0;j<col2;j++){
        printf(" %f",z[i][j]);
        }
        printf("\n");
    }
return 0;
}