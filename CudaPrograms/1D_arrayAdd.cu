#include "hip/hip_runtime.h"
%%cu
//add two 1D matrix 
// here we launch one single block which has user defined threads(let's threads =10)


#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void vectorAdd(float *x,float *y, float *z,int threads){
    int id= threadIdx.x;
    if(id<threads)
        z[id]=x[id]+y[id];
    else
        printf("You launch more than given threads: ");
}

int main(int *argc,char **argv[]){
    int threads=10;   //this value is user defined

    float x[threads]={1,2,3,4,5,6,7,8,9,10};
    float y[threads]={1,2,3,4,5,6,7,8,9,10};
    float z[threads];
    
    //variables for gpu
    float *d_x;
    float *d_y;
    float *d_z;

    //now we allocate space in gpu memory
    int array_size=threads*sizeof(float);

    hipMalloc( (void**)&d_x,array_size);
    hipMalloc( (void**)&d_y,array_size);
    hipMalloc( (void**)&d_z,array_size);

    //now we copy these vector(array) in gpu memory
    hipMemcpy( d_x, x , array_size , hipMemcpyHostToDevice);
    hipMemcpy( d_y, y , array_size , hipMemcpyHostToDevice);

    // now we launch kernel
    vectorAdd<<<1,threads>>>(d_x,d_y,d_z,threads);

    // copy array from gpu to cpu memory
    hipMemcpy( z , d_z , array_size , hipMemcpyDeviceToHost);

    hipFree( d_x);
    hipFree( d_y);
    hipFree( d_z);


    // print the values of output array
    printf("values of array z:\n");
    for(int j=0;j<threads;j++){
        printf(" %f",z[j]);
    }
return 0;
}